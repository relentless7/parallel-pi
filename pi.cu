#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include "pi.h"
#include <math.h>
#include <random>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>

//Kernel taken from  OakRidge Labs, what it does it pretty straight forward...
__global__ void throwdarts_kernel(int* hits, float* randomnums)
{
	int i;
	double x,y,z;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	i = tid;
	int xidx = 0, yidx = 0;

	xidx = (i+i);
	yidx = (xidx+1);

	x = randomnums[xidx];
	y = randomnums[yidx];
	z = ((x*x)+(y*y));

	if (z<=1)
		hits[tid] = 1;
	else
		hits[tid] = 0;	
}

void CUDAErrorCheck()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{	
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

#define NUM_THREADS 1024

//wrapper for CUDA calls
std::int64_t ThrowDarts(std::int64_t num_trials)
{
	int device_num;
	size_t mem_free = 0;
	size_t mem_tot = 0;
	
	hipSetDevice(0);
	hipGetDevice(&device_num);
	hipDeviceReset();
	hipMemGetInfo  (&mem_free, &mem_tot);
	//std::cout<<"Free memory before copy dev: "<<mem_free<<" Total: "<<  mem_tot <<std::endl;	

	std::int64_t num_trials_remaining = num_trials; 
	float *randomnums;
		
	std::int64_t max_mem_per_iter = mem_free - 30000000; //magic number to leave enough buffer so we don't run out of memory.
	std::int64_t max_trials_per_iter = max_mem_per_iter/(2 * sizeof(float) + sizeof(int));	

	//we have to promote up to 64-bit math to do this roundup(uint/uint) = uint + uint - 1/uint is not safe at max values
	//instead double holds a uint max size variable, we will use ceil and round up.
	std::int64_t number_of_iters = ceil((double)num_trials_remaining/(double)max_trials_per_iter);

	//std::cout << "Max Trials per iteration: " << max_trials_per_iter << std::endl;	
	//std::cout << "Number of kernel launches needed: " << number_of_iters << std::endl;
	std::int64_t reducedcount = 0;

	int blocks = (max_trials_per_iter + NUM_THREADS - 1) / NUM_THREADS;
	int threads = NUM_THREADS;

	hipMalloc((void**)&randomnums, (2 * max_trials_per_iter) * sizeof(float));
	CUDAErrorCheck();

	int* device_hits;
	int hit_count;

	hipMalloc((void**)&device_hits, max_trials_per_iter * sizeof(int));
	CUDAErrorCheck();

	while(num_trials_remaining > 0)
	{
		std::int64_t num_trials_to_launch = min(num_trials_remaining, max_trials_per_iter);
		num_trials_remaining -= num_trials_to_launch;
	
		// Use CuRand to generate an array of random numbers on the device
		int status;
		hiprandGenerator_t gen;
		status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
		status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
		status |= hiprandGenerateUniform(gen, randomnums, (2*num_trials_to_launch));
		status |= hiprandDestroyGenerator(gen);
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			printf("CuRand Failure\n");
			exit(EXIT_FAILURE);
		}

		blocks = (num_trials_to_launch + threads - 1) / threads;	

		//one dart throw per thread
		throwdarts_kernel <<<blocks, threads>>> (device_hits, randomnums);
		hipDeviceSynchronize();
		CUDAErrorCheck();
		
		//leave the results on the device and do an inclusive scan (rather than count them on the host)
		thrust::inclusive_scan(thrust::device, device_hits, device_hits + num_trials_to_launch, device_hits);
		//mem copy the last element, it contains the total hit count
		hipMemcpy(&hit_count, device_hits + num_trials_to_launch-1, sizeof(int), hipMemcpyDeviceToHost);
		
		reducedcount = reducedcount + hit_count;

	}

	hipFree(randomnums);
	hipFree(device_hits);

	return reducedcount; 
}

double GPUMonteCarloPi::CalculateApproximation(std::int64_t num_trials)
{
	return ((double)ThrowDarts(num_trials)/(double)num_trials) * 4.0;
}

double HybridMonteCarloPi::CalculateApproximation(std::int64_t num_trials)
{
	std::int64_t gpu_hits = 0;
	std::int64_t cpu_hits = 0;
	std::int64_t num_gpu_trials = (std::int64_t)(num_trials * gpu_ratio);
	std::int64_t num_cpu_trials = num_trials - num_gpu_trials;

    #pragma omp parallel
    {
		#pragma omp single nowait
		{
			gpu_hits = ThrowDarts(num_gpu_trials);
		}
				
		//this blob is not a a static function because getting cpu_hits to remain public in that context
		//is too difficult, so we will deal with a tiny bit of bloat.
		std::random_device rseed;
		std::mt19937 rgen(rseed()); // mersenne_twister
		std::uniform_real_distribution<double> rdist(0.0,1.0);
					
		#pragma omp for schedule(guided) reduction(+:cpu_hits)
		for(std::int64_t i=0; i < num_cpu_trials; ++i)
		{
			double random_x = rdist(rgen);
			double random_y = rdist(rgen);
			//if we are in the circle increment the count.
			if(random_x*random_x + random_y*random_y <= 1.0)
			{
				cpu_hits = cpu_hits + 1; 
			}
		}
    }
	//largest double can actually hold the largest int by standard
	return (double)(cpu_hits + gpu_hits)/(double)num_trials * 4.0;
}
